#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif

#include <Exceptions.h>
#include <ImageIO.h>
#include <ImagesCPU.h>
#include <ImagesNPP.h>

#include <string.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <npp.h>

#include <hip/hip_runtime_api.h>
#include <helper_string.h>

// #include <opencv2/opencv.hpp>
#include <hipfft/hipfft.h>

#include ""
#include <helper_functions.h>
#include <cmath>

#include <iostream>
#include <hipfft/hipfft.h>

#include <cstdlib> // For system()
#include <cstring> // Add this line to include the <cstring> header

#define prLN std::cout << "Made it to line number: " << __LINE__ << std::endl;

using namespace std;

__global__ void applyFilter(hipfftComplex *d_freq, const hipfftComplex *filter)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Multiply the real and imaginary parts of FFT data by the corresponding parts of the filter
  d_freq[idx].x *= filter[idx].x;
  d_freq[idx].y *= filter[idx].x; // Using filter[idx].x since filter[idx].y is expected to be 0
}

void writeComplexToFile(const hipfftComplex *signal, int N, const std::string &fileName)
{
  // Open the file for writing
  std::ofstream outFile(fileName);

  // Check if the file was opened successfully
  if (!outFile.is_open())
  {
    std::cerr << "Error: Unable to open file " << fileName << " for writing." << std::endl;
    return;
  }

  for (int i = 0; i < N; ++i)
  {
    outFile << signal[i].x << " " << signal[i].y << std::endl;
  }
  outFile.close();

}

void TopHat(int width, hipfftComplex *h_data, int N)
{
  for (int i = 0; i < N; ++i)
  {
    if (i >= N / 2 - width / 2 && i < N / 2 + width / 2)
    {
      h_data[i].x = 1;
      h_data[i].y = 0;
    }
    else
    {
      h_data[i].x = 0;
      h_data[i].y = 0;
    }
  }
}

  int main(int argc, char *argv[])
  {
    printf("%s Starting...\n\n", argv[0]);

    try
    {

      // ************ FFT processing *************************

      const int N = 40; // all test have same spatial grid axis

      //Allocate memory for the input data and copy to device
      hipfftComplex *h_data = new hipfftComplex[N];
      hipfftComplex *d_data;
      hipMalloc((void **)&d_data, N * sizeof(hipfftComplex));
      std::string label = "xx";

            // Create a 1D FFT plan
      hipfftHandle plan;
      hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

      hipfftComplex *result = new hipfftComplex[N];

      std::string command = "XX";

      int f = 0;


      /* *****************************************************
       ******************** TEST freq = 1 ***************************
       **************************************************** */
      // Test1: Explore FFT output for known test case.
      // By design, f(x_j,p) is exp(2*PI*i*(f/N)*j)
      // Only non zero freq will be j=f
      // f gets redefined for each test
      // f is the number of cycles over the span
      // N is the same for all test cases

      f = 1; // Lambda = N/f = 40/1 = 40 <== 1 cycle over the range

      // Define the input data on the host
      for (int i = 0; i < N; ++i)
      {
        h_data[i].x = cos(-2 * M_PI * f * i / N);
        h_data[i].y = sin(-2 * M_PI * f * i / N);
      }

      //prLN; // prints the current line number. Defined using #define

      // need to save complex array to file in order to plot with gnuplot
      // unable to figure out how to use gnuplot without saving to a file.
      writeComplexToFile(h_data, N, "temp.dat");

      // gp_script accepts 5 arguments
      command = "gnuplot -c gp_script.gp 'f=1 Input Data:Lambda = 40' 'index' 'amp' 'temp.dat' 'plot_f01a_InputData.png'";
      system(command.c_str());

      // copy h_data to data on GPU
      hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

      // Perform the FFT
      // plan already defined at top
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "temp.dat");

      command = "gnuplot -c gp_script.gp 'f=1 FFT of Input Data: Lambda=40' 'Cycles over range:freq' 'amp' 'temp.dat' 'plot_f01b_FFTofData.png'";
      system(command.c_str());

      /* *****************************************************
       ******************** TEST freq = 20********************
       **************************************************** */
      f = 20; // Lambda = N/p = 40/20 = 2 <== 1

      // Define the input data on the host
      for (int i = 0; i < N; ++i)
      {
        h_data[i].x = cos(-2 * M_PI * f * i / N);
        h_data[i].y = sin(-2 * M_PI * f * i / N);
      }

      writeComplexToFile(h_data, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=20 Input Data' 'index' 'amp' 'file_input_data.dat' 'plot_f20a_InputData.png'";
      system(command.c_str());

      hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=20 FFT Input Data' 'Cycles over range:freq' 'amp' 'file_input_data.dat' 'plot_f20b_FFTofData.png'";
      system(command.c_str());

      // Perform the inverse FFT
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write invFFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=20 invFFT' 'index' 'amp' 'file_input_data.dat' 'plot_f20c_invFFT.png'";
      system(command.c_str());

      /* *****************************************************
       ******************** TEST freq = 15********************
       **************************************************** */
      f = 15; // Lambda = N/p = 40/20 = 2 <== 1

      // Define the input data on the host
      for (int i = 0; i < N; ++i)
      {
        h_data[i].x = cos(-2 * M_PI * f * i / N);
        h_data[i].y = sin(-2 * M_PI * f * i / N);
      }

      writeComplexToFile(h_data, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=15 Input Data' 'index' 'amp' 'file_input_data.dat' 'plot_f15a_InputData.png'";
      system(command.c_str());

      hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=15 FFT Input Data' 'freq' 'amp' 'file_input_data.dat' 'plot_f15b_FFTofData.png'";
      system(command.c_str());

      // Perform the inverse FFT
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write invFFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=15 invFFT' 'index' 'amp' 'file_input_data.dat' 'plot_f15c_invFFT.png'";
      system(command.c_str());

      /* *****************************************************
       ******************** TEST Top Hat***************************
       **************************************************** */

      // want to also create the freq filter
      // Define the input data on the host
      // Data is a top hat source
      for (int i = 0; i < N; ++i)
      {
        if (i > 9 && i < 31 )
        {
          h_data[i].x = 1;
          h_data[i].y = 0;
        } else
        {
          h_data[i].x = 0;
          h_data[i].y = 0;
        } 
      }

      

            // Write TopHat data to a file and create png
      writeComplexToFile(h_data, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'Top Hat' 'index' 'amp' 'file_input_data.dat' 'plot_fTopHata_InputData.png'";
      system(command.c_str());

      hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

      // Perform the FFT
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'freq=20 FFT Input Data' 'freq' 'amp' 'file_input_data.dat' 'plot_fTopHatb_FFT.png'";
      system(command.c_str());

      // Perform the inverse FFT
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      hipfftComplex *result_norm = new hipfftComplex[N];
      for (int i = 0; i < N; i++)
      {
        result_norm[i].x = result[i].x / N;
        result_norm[i].y = result[i].y / N;
      }

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");
      command = "gnuplot -c gp_script.gp 'freq=20 invFFT' 'index' 'amp' 'file_input_data.dat' 'plot_fTopHatc_invFFT.png'";
      system(command.c_str());

      //Let's redo but chop out the higher frequencies before the invFFT
      //will need to reset d_data to h_data

      hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

      // Perform the FFT. d_data will contains the unfiltered fft
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

      // Let's chop higher frequencies out of FFT
      hipfftComplex h_filter[N];

      // Initialize filter array
      for (int i = 0; i < N; ++i)
      {
        if (i <= 5 || i >= 35)
        {
          h_filter[i].x = 1.0f;
          h_filter[i].y = 0.0f;
        }
        else
        {
          h_filter[i].x = 0.0f;
          h_filter[i].y = 0.0f;
        }
      }

      // Copy filter array to device memory
      hipfftComplex *d_filter;
      hipMalloc(&d_filter, N * sizeof(hipfftComplex));
      hipMemcpy(d_filter, h_filter, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

      // Define grid and block sizes
      //int blockSize = 256;
      //int numBlocks = (N + blockSize - 1) / blockSize;

      int threadsPerBlock = 256;
      int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

      //vectorMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

      // Call the kernel function
      applyFilter<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_filter);

      //let's now copy back the filterred spectrum
      //d_data has been modified..let's copy and look

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT filtered data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");

      command = "gnuplot -c gp_script.gp 'TopHat filtered freq spectrum' 'freq' 'amp' 'file_input_data.dat' 'plot_fTopHatd_filteredFFT.png'";
      system(command.c_str());

      // Perform the inverse FFT
      hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

      // Copy the result back to host memory
      hipMemcpy(result, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(result, N, "file_input_data.dat");
      command = "gnuplot -c gp_script.gp 'Top Hat filtered InvFFT ' 'index' 'amp' 'file_input_data.dat' 'plot_fTopHate_filteredInvFFT.png'";
      system(command.c_str());



      // Check for kernel launch errors
      hipError_t launchError = hipGetLastError();
      if (launchError != hipSuccess)
      {
        printf("Kernel launch error: %s\n", hipGetErrorString(launchError));
        // Further error handling if needed
      }

      /* *****************************************************
       ******************** TEST Many FFT***************************
       ******************** Do FFT 2 TopHat signals at once **********
       **************************************************** */

      // Define and populate topHat_30 array
      hipfftComplex topHat_30[N];
      TopHat(30, topHat_30, N);

      // Define and populate topHat_20 array
      hipfftComplex topHat_20[N];
      TopHat(20, topHat_20, N);

      // Calculate total size for the large array
      const int totalSize = 2 * N;
      hipfftComplex largeArray[totalSize];

      // Copy topHat_30 array into the beginning of largeArray
      std::memcpy(largeArray, topHat_30, N * sizeof(hipfftComplex));

      // Copy topHat_20 array into the remaining space of largeArray
      std::memcpy(largeArray + N, topHat_20, N * sizeof(hipfftComplex));

      // Write TopHat largeArray data to a file and create png
      writeComplexToFile(largeArray, totalSize, "file_input_data.dat");
      command = "gnuplot -c gp_script.gp 'topHat=30 & topHat=20 together ' 'index' 'amp' 'file_input_data.dat' 'plot_ManyFFTa_InputData.png'";
      system(command.c_str());

      // Allocate memory for d_largeArray on the device
      hipfftComplex *d_largeArray;
      hipMalloc(&d_largeArray, 2 * N * sizeof(hipfftComplex));

      // Copy largeArray to d_largeArray on the device
      hipMemcpy(d_largeArray, largeArray, 2 * N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

      const int batch = 2;              // Number of FFTs to perform
      
      // Create a hipfftHandle for FFT plan
      int n[1] = {N}; // Size of each dimension of the input data
      hipfftHandle planM;
      hipfftPlanMany(&planM, 1, n, nullptr, 1, N, nullptr, 1, N, HIPFFT_C2C, batch); // Create a 1D complex-to-complex FFT plan for a batch of signals

      // Execute FFT on the batch of signals
      hipfftExecC2C(planM, d_largeArray, d_largeArray, HIPFFT_FORWARD); // Forward FFT

      hipfftComplex *resultMany = new hipfftComplex[totalSize];

      // Copy the result back to host memory
      hipMemcpy(resultMany, d_largeArray, totalSize * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

      // Write FFT data to a file and create png
      writeComplexToFile(resultMany, totalSize, "file_input_data.dat");
      command = "gnuplot -c gp_script.gp 'Top Hat Many FFT ' 'freq every 40' 'amp' 'file_input_data.dat' 'plot_ManyFFTb_ManyFFT.png'";
      system(command.c_str());

      // Free memory and destroy plan
      delete[] result, result_norm, h_data;
      hipFree(d_data);
      hipFree(d_filter);
      hipfftDestroy(plan);
      hipfftDestroy(planM);
      hipFree(d_largeArray);

      // ************end FFT processing ******************

      exit(EXIT_SUCCESS);
    }
  catch (npp::Exception &rException)
  {
    std::cerr << "Program error! The following exception occurred: \n";
    std::cerr << rException << std::endl;
    std::cerr << "Aborting." << std::endl;

    exit(EXIT_FAILURE);
  }
  catch (...)
  {
    std::cerr << "Program error! An unknow type of exception occurred. \n";
    std::cerr << "Aborting." << std::endl;

    exit(EXIT_FAILURE);
    return -1;
  }

  return 0;
}
